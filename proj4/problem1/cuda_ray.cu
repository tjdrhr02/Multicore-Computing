
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define SPHERES 20

#define rnd( x ) (x * rand() / RAND_MAX)
#define INF 2e10f
#define DIM 2048

struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;
    __device__ float hit(float ox, float oy, float* n) {
        float dx = ox - x;
        float dy = oy - y;
        if (dx * dx + dy * dy < radius * radius) {
            float dz = sqrtf(radius * radius - dx * dx - dy * dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

// using CUDA
__global__ void kernel (Sphere* s, unsigned char* ptr)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
 
    int offset = x + y * DIM;
    float ox = (x - DIM/2);
    float oy = (y - DIM/2);
 
    //printf("x:%d, y:%d, ox:%f, oy:%f\n",x,y,ox,oy);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;
    for(int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }

    ptr[offset * 4 + 0] = (int)(r * 255);
    ptr[offset * 4 + 1] = (int)(g * 255);
    ptr[offset * 4 + 2] = (int)(b * 255);
    ptr[offset * 4 + 3] = 255;
}

void ppm_write(unsigned char* bitmap, int xdim, int ydim, FILE* fp)
{
    int i, x, y;
    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");
    for (y = 0; y < ydim; y++) {
        for (x = 0; x < xdim; x++) {
            i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4 * i], bitmap[4 * i + 1], bitmap[4 * i + 2]);
        }
        fprintf(fp, "\n");
    }
}


int main(int argc, char* argv[])
{
    unsigned char* bitmap;
    Sphere* dev_s;
    unsigned char* dev_b;

    srand(time(NULL));
    FILE* fp = fopen("result.ppm", "w");

    Sphere* temp_s = (Sphere*)malloc(sizeof(Sphere) * SPHERES); 
    for (int i = 0; i < SPHERES; i++) {
        temp_s[i].r = rnd(1.0f);
        temp_s[i].g = rnd(1.0f);
        temp_s[i].b = rnd(1.0f);
        temp_s[i].x = rnd(2000.0f) - 1000;
        temp_s[i].y = rnd(2000.0f) - 1000;
        temp_s[i].z = rnd(2000.0f) - 1000;
        temp_s[i].radius = rnd(200.0f) + 40;
    }
 
    hipMalloc((void**)&dev_s, SPHERES * sizeof(Sphere));
    hipMalloc((void**)&dev_b, sizeof(unsigned char) * DIM * DIM * 4);
 
    hipMemcpy(dev_s, temp_s, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice);

    clock_t start_time = clock();

    // number of blocks, threads per block
    // it computes using CUDA and get output of ppm image
    kernel<<<dim3((DIM + 15) / 16, (DIM + 15) / 16), dim3(16, 16)>>>(dev_s, dev_b);
 
    bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM * DIM * 4);
    hipMemcpy(bitmap, dev_b, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost);

    clock_t end_time = clock();
    clock_t diff_time = end_time - start_time;
	  printf("CUDA ray tracing: %f sec. \n", (double)diff_time/CLOCKS_PER_SEC);
 
    ppm_write(bitmap, DIM, DIM, fp);
    printf("[%s] was generated.\n", "result.ppm");
 
    fclose(fp);
    free(bitmap);
    free(temp_s);
    hipFree(dev_s); hipFree(dev_b);
    return 0;
}
