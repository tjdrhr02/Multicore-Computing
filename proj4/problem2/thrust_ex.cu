#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <thrust/transform.h>
#include <thrust/sequence.h>

#include <stdio.h>
#include <time.h>
#include <iostream>

long num_steps = 200000; 
double step = 1.0/(double) num_steps;

template<typename T>
struct integral
{
    double step;

    integral(double step) : step(step){}

    __host__ __device__
        T operator()(const T &i) const {
            double x = (i+0.5)*step;
            return 4.0/(1.0+x*x);
        }
};

int main(){
    clock_t start_time = clock();
    thrust::device_vector<int> index(num_steps);

    // make new sequence
    thrust::sequence(index.begin(), index.end());

    integral<double> unary_op(step);
    thrust::plus<double> binary_op;
    double init = 0.0;
    
    // transform (using function unary_op, set by 'integral') and reduction (reduce to a single value, set by summation)
    double sum = thrust::transform_reduce(index.begin(), index.end(), unary_op, init, binary_op);
    double pi = step * sum;
    printf("pi=%.10lf\n",pi);

    clock_t end_time = clock();

    clock_t diff_time = end_time - start_time;
	  printf("execution time: %.3lf sec. \n", (double)diff_time/CLOCKS_PER_SEC);

    return 0;
}
